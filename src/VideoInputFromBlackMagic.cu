#include "hip/hip_runtime.h"
#include "VideoInputFromBlackMagic.hpp"

// usleep
#include <unistd.h>

#include <stdio.h>

__global__ void separateFramesKernel(unsigned char* bothFrames, unsigned char* rightFrame, unsigned char* leftFrame, int sizeRow){

   const int outputXIndex = blockIdx.x * blockDim.x + threadIdx.x;
   const int outputYIndex = blockIdx.y * blockDim.y + threadIdx.y;

   int output_tid  = outputYIndex * sizeRow + (outputXIndex * 3);
   int output_tid_p1  = (outputYIndex+1) * sizeRow + ((outputXIndex) * 3);
   int output_tid_m1  = (outputYIndex-1) * sizeRow + ((outputXIndex) * 3);

   if(outputYIndex % 2 == 0){
      rightFrame[output_tid] = bothFrames[output_tid];
      rightFrame[output_tid+1] = bothFrames[output_tid+1];
      rightFrame[output_tid+2] = bothFrames[output_tid+2];
      rightFrame[output_tid_p1] = bothFrames[output_tid];
      rightFrame[output_tid_p1+1] = bothFrames[output_tid+1];
      rightFrame[output_tid_p1+2] = bothFrames[output_tid+2];

   } else {
      leftFrame[output_tid_m1] = bothFrames[output_tid];
      leftFrame[output_tid_m1+1] = bothFrames[output_tid+1];
      leftFrame[output_tid_m1+2] = bothFrames[output_tid+2];
      leftFrame[output_tid] = bothFrames[output_tid];
      leftFrame[output_tid+1] = bothFrames[output_tid+1];
      leftFrame[output_tid+2] = bothFrames[output_tid+2];
   }

}

bool VideoInputFromBlackMagic::separateFrames(cv::Mat* left, cv::Mat* right, cv::Mat* combined){

   if(!this->initCuda){
      //Init the array
      this->sizeCurrentImageData = combined->rows * combined->step;
      this->sizeImageRData = right->rows * right->step;
      this->sizeImageLData = left->rows * left->step;
      hipMalloc((void **) &this->currentImageDevice, this->sizeCurrentImageData);
      hipMalloc((void **) &this->ImageRightDevice, this->sizeImageRData);
      hipMalloc((void **) &this->ImageLeftDevice, this->sizeImageLData);
      this->initCuda = true;
   }

   hipMemcpy(this->currentImageDevice, combined->ptr(), this->sizeCurrentImageData, hipMemcpyHostToDevice) ;
   hipMemcpy(this->ImageRightDevice, right->ptr(), this->sizeImageRData, hipMemcpyHostToDevice) ;
   hipMemcpy(this->ImageLeftDevice, left->ptr(), this->sizeImageLData, hipMemcpyHostToDevice) ;

   //Specify a reasonable block size
   const dim3 block(16,16);

   //Grid
   const dim3 grid((combined->cols + block.x - 1)/block.x, (combined->rows + block.y - 1)/block.y);

   //Call the kernel
   separateFramesKernel<<<grid,block>>>(this->currentImageDevice, this->ImageRightDevice, this->ImageLeftDevice, combined->step);

   hipDeviceSynchronize();
   hipMemcpy(right->ptr(),this->ImageRightDevice,this->sizeImageRData,hipMemcpyDeviceToHost);
   hipMemcpy(left->ptr(),this->ImageLeftDevice,this->sizeImageLData,hipMemcpyDeviceToHost);

   return true;
}

//Constructor
VideoInputFromBlackMagic::VideoInputFromBlackMagic(): m_refCount(1){

   this->running = false;
   this->initialized = false;
   this->initCuda = false;

}

//Destructor
//TODO FREE CUDA MEMORY
VideoInputFromBlackMagic::~VideoInputFromBlackMagic(){

}

//Run call
std::thread VideoInputFromBlackMagic::run(){
   printf("VideoInputFromBlackMagic : run function has been called...\n");
   std::thread mainThread(runThread, this);
   return mainThread;
}

//Run sub function
void VideoInputFromBlackMagic::runThread(VideoInputFromBlackMagic* context){
   context->runInput();
}

//Run (real stuff)
void VideoInputFromBlackMagic::runInput(){

   fprintf(stdout, "Run\n");

   if(!this->running){

      this->running=true;

      int idx;

      //Check result
      HRESULT result;

      IDeckLink* deckLink = NULL;
      IDeckLinkInput* g_deckLinkInput = NULL;
      IDeckLinkAttributes* deckLinkAttributes = NULL;
      IDeckLinkIterator* deckLinkIterator = CreateDeckLinkIteratorInstance();
      IDeckLinkDisplayModeIterator* displayModeIterator = NULL;
      IDeckLinkDisplayMode* displayMode = NULL;

      char* displayModeName = NULL;
      BMDDisplayModeSupport displayModeSupported;
      bool formatDetectionSupported;

      if (!deckLinkIterator)
      {
         fprintf(stderr, "This application requires the DeckLink drivers installed.\n");
         return;
      } 

      //Get the DeckLink Inputs
      result = deckLinkIterator->Next(&deckLink);
      result = deckLink->QueryInterface(IID_IDeckLinkInput, (void**)&g_deckLinkInput);

      if(result != S_OK){
         fprintf(stdout, "Cannot get the Input : DeckLink Error\n");
         return;
      } 

      //Get the DeckLink attributes (that may not correctly work: format detection does not properly work)
      result = deckLink->QueryInterface(IID_IDeckLinkAttributes, (void**)&deckLinkAttributes);

      if (!(result == S_OK)){
         fprintf(stdout, "Cannot get the DeckLink attributes : DeckLink Error\n");
         return;
      }

      //Format detection
      result = deckLinkAttributes->GetFlag(BMDDeckLinkSupportsInputFormatDetection, &formatDetectionSupported);
      if (result != S_OK || !formatDetectionSupported){
         fprintf(stdout,"Cannot get the format input: DeckLink Error\n");
         return;
      } 

      //Index for the different inputs
      idx = 0;

      //Get all the displayModes
      result = g_deckLinkInput->GetDisplayModeIterator(&displayModeIterator);

      if (result != S_OK){
         fprintf(stdout,"Cannot set an iterator on the different display modes: DeckLink Error\n");
      }

      //Set idx
      while ((result = displayModeIterator->Next(&displayMode)) == S_OK)
      {

         if (idx == 0)
            break;
         --idx;   
         displayMode->Release();

      }

      if (result != S_OK || displayMode == NULL){

         fprintf(stdout,"Cannot get the main display mode: DeckLink Error\n");
         return;

      } 

      //Get Mode name: useless
      result = displayMode->GetName((const char**)&displayModeName);

      // Check display mode is supported with given options
      result = g_deckLinkInput->DoesSupportVideoMode(bmdModeHD1080p30, bmdFormat8BitYUV, bmdDisplayModeColorspaceRec709, &displayModeSupported, NULL);

      if (result != S_OK){
         fprintf(stdout,"Video Mode not supported : aborted\n");
         return;
      } 


      if (displayModeSupported == bmdDisplayModeNotSupported)
      {
         fprintf(stdout, "The display mode %s is not supported with the selected pixel format\n", displayModeName);
         return;

      } 

      //Set the callback on this ( will defined callback on VideoInputFrameArrived and others functions when images arrives or when other events happens
      g_deckLinkInput->SetCallback(this);

      //Enable the video input with the selected format
      result = g_deckLinkInput->EnableVideoInput(bmdModeHD1080p30, bmdFormat8BitYUV, bmdDisplayModeColorspaceRec709);


      if (result != S_OK)
      {
         fprintf(stderr, "Failed to enable video input. Maybe another application is using the card.\n");
         return;

      } 

      //Disable the audio
      result = g_deckLinkInput->DisableAudioInput();

      //Start the stream
      result = g_deckLinkInput->StartStreams();
      if (result != S_OK){
         fprintf(stdout,"Error while starting the streaming : aborted\n");
      }

      while(this->running){
         //Nothing thread must not end... this is dirty TODO mutex?
      }

   }

}

//A frame arrived
HRESULT VideoInputFromBlackMagic::VideoInputFrameArrived(IDeckLinkVideoInputFrame* videoFrame, IDeckLinkAudioInputPacket* audioFrame){

   //Here a good idea can be to ignore frames sometimes..

   if (!videoFrame){

      fprintf(stdout,"Update: No video frame\n");
      return S_FALSE;

   } 

   void* data;

   if (FAILED(videoFrame->GetBytes(&data))){
      fprintf(stdout,"Fail obtaining the data from videoFrame\n");
      return S_FALSE;
   }

   cv::Mat loadedImage;
   cv::Mat mat = cv::Mat(videoFrame->GetHeight(), videoFrame->GetWidth(), CV_8UC2, data, videoFrame->GetRowBytes());
   cv::cvtColor(mat, loadedImage, CV_YUV2BGR_UYVY);
   cv::Mat loadedImageRight = cv::Mat::zeros(loadedImage.rows,loadedImage.cols, loadedImage.type());
   cv::Mat loadedImageLeft = cv::Mat::zeros(loadedImage.rows,loadedImage.cols, loadedImage.type()) ;

   if (!loadedImage.data){
      fprintf(stdout,"No frame loaded from the video : mainImage will not be updated\n");
   } else {

      if(!this->separateFrames(&loadedImageLeft, &loadedImageRight, &loadedImage)){
         fprintf(stdout,"Error while the separation of left and right frame\n");
      }

      //Update the images
      //Mutex here
      this->mtxImages.lock();
      this->currentImageLeft = loadedImageLeft.clone();
      this->currentImageRight = loadedImageRight.clone();
      this->initialized = true;
      this->mtxImages.unlock();

   }

   return S_OK;

}

//DeckLink stuff: not important
ULONG VideoInputFromBlackMagic::AddRef(void)
{
   return __sync_add_and_fetch(&m_refCount, 1);
}

ULONG VideoInputFromBlackMagic::Release(void)
{
   int32_t newRefValue = __sync_sub_and_fetch(&m_refCount, 1);
   if (newRefValue == 0)
   {
      return 0;
   }
   return newRefValue;
}

HRESULT VideoInputFromBlackMagic::VideoInputFormatChanged(BMDVideoInputFormatChangedEvents events, IDeckLinkDisplayMode *mode, BMDDetectedVideoInputFormatFlags formatFlags){

   fprintf(stdout,"VideoInputFormatChanged: Not supported\n");   
   return S_OK;

}

//Stop TODO unlock mutex
void VideoInputFromBlackMagic::stop(){
   this->running=false;
}

bool VideoInputFromBlackMagic::isRunning(){
   return this->running;
}

bool VideoInputFromBlackMagic::isInitialized(){
   return this->initialized;
}

void VideoInputFromBlackMagic::getFrames(cv::Mat & leftI, cv::Mat & rightI){

   this->mtxImages.lock();
   leftI = this->currentImageLeft.clone();
   rightI = this->currentImageRight.clone();
   this->mtxImages.unlock();

}
